#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
using namespace std;

__global__ void sum_vect(int *a, int *b,int n)
{
    __shared__ int arr[10];
    int k = threadIdx.x;
    arr[k] = a[k];
    __syncthreads();
 
    int sum = 0;
    for(int i=0;i<n;i++)
      sum += arr[k];
    *b = sum / n;
}

int main()
{
    int *a = new int[10];
    int *b = new int(0);
 
    for(int i=0;i<10;i++)
      a[i] = (i+1);

    int *devc_a, *devc_b;

    hipMalloc(&devc_a, 10 * sizeof(int));
    hipMalloc(&devc_b, sizeof(int));
 
    hipMemcpy(devc_a, a, 10 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devc_b, b, sizeof(int), hipMemcpyHostToDevice);
 
    sum_vect<<<1,10>>>(devc_a,devc_b,10);

    hipMemcpy(b, devc_b,sizeof(int), hipMemcpyDeviceToHost);
 
    cout<<"Average: "<<*b;
 
    hipFree(devc_a);
    hipFree(devc_b);
    delete(a);
    delete(b);
    return 0;
}