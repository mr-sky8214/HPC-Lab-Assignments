#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
using namespace std;

__global__ void swap_nums(int *a, int *b)
{
    int tmp = *a;
    *a = *b;
    *b = tmp;
}

int main()
{
    int *a = new int(5);
    int *b = new int(10);
 
     cout<<"Before swapping "<<*a<<" "<<*b<<endl;
    int *devc_a, *devc_b;

    hipMalloc(&devc_a, sizeof(int));
    hipMalloc(&devc_b, sizeof(int));
 
    hipMemcpy(devc_a, a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devc_b, b, sizeof(int), hipMemcpyHostToDevice);
 
    swap_nums<<<1,1>>>(devc_a,devc_b);

    hipMemcpy(a, devc_a,sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(b, devc_b,sizeof(int), hipMemcpyDeviceToHost);
 
    cout<<"After swapping "<<*a<<" "<<*b;
 
    hipFree(devc_a);
    hipFree(devc_b);
    delete(a);
    delete(b);
    
    return 0;
}