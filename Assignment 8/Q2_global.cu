#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
using namespace std;

__device__ int global_arr[10];

__device__ void initialize(int i)
{
    global_arr[i] = i + 1;
}

__global__ void getArr(int *a)
{
    int idx = threadIdx.x;
    initialize(idx);
    a[idx] = global_arr[idx];
}

int main()
{
    int *a = new int[10], *devc_a;
    hipMalloc(&devc_a, 10 * sizeof(int));
    getArr<<<1,10>>>(devc_a);
    hipMemcpy(a, devc_a, 10 * sizeof(int), hipMemcpyDeviceToHost);
 
    cout<<"Copied array: ";
    for(int i=0;i<10;i++)
      cout<<a[i]<<" ";

    hipFree(devc_a);
    delete(a);
    return 0;
}
