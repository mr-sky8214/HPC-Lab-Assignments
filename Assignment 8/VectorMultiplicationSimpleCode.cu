#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
using namespace std;

const int r1 = 20, c1 = 30, r2 = 30, c2 = 20;

__global__ void mulMatrix(int *a, int * b, int *c)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
 
    int id_c = c2 * y + x;

    c[id_c] = 0;
    for(int k = 0 ; k < c1; k++)
    {
        int id_a = c1 * y + k;
        int id_b = c2 * k + x;
        c[id_c] = c[id_c] + a[id_a] * b[id_b];
    }
}


int main()
{
    int host_a[r1][c1] , host_b[r2][c2], host_c[r1][c2];
    int *devc_a, *devc_b, *devc_c;
 
    hipMalloc(&devc_a, r1 * c1 * sizeof(int));
    hipMalloc(&devc_b, r2 * c2 * sizeof(int));
    hipMalloc(&devc_c, r1 * c2 * sizeof(int));
 
    for(int i=0;i<r1;i++)
    {
        for(int j=0;j<c1;j++)
          host_a[i][j] = i + j + 1;
    }
 
    for(int i=0;i<r2;i++)
    {
        for(int j=0;j<c2;j++)
          host_b[i][j] = (i + 1) * (j + 1);
    }
 
    hipMemcpy(devc_a, host_a, r1 * c1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devc_b, host_b, r2 * c2 * sizeof(int), hipMemcpyHostToDevice);
 
    dim3 grid(c2,r1);

    mulMatrix<<<grid, 1>>>(devc_a, devc_b, devc_c);

    hipMemcpy(host_c, devc_c, r1 * c2 * sizeof(int), hipMemcpyDeviceToHost);
 
    for(int i=0;i<r1;i++)
    {
        for(int j=0;j<c2;j++)
          cout<<host_c[i][j]<<" ";
        cout<<endl;
    }

    return 0;
}