#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
using namespace std;

__global__ void addVectors(int *a, int *b, int *c)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    c[id] = a[id] + b[id];
}

int main()
{
    int n = 1e5, i = 0;
    int *host_a, *host_b, *host_c;
    int *devc_a, *devc_b, *devc_c;
 
    host_a = (int *)malloc(n * sizeof(int));
    host_b = (int *)malloc(n * sizeof(int));
    host_c = (int *)malloc(n * sizeof(int));
 
    hipMalloc(&devc_a, n * sizeof(int));
    hipMalloc(&devc_b, n * sizeof(int));
    hipMalloc(&devc_c, n * sizeof(int));
 
    for(i = 0; i < n; i++)
    {
        host_a[i] = i;
        host_b[i] = log(i)/log(2);
    }
 
    hipMemcpy(devc_a, host_a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(devc_b, host_b, n * sizeof(int), hipMemcpyHostToDevice);
 
    int blocks = 1000, threads;
    threads = n / blocks;

    addVectors<<<blocks,threads>>>(devc_a,devc_b,devc_c);
 
    hipMemcpy(host_c, devc_c, n * sizeof(int), hipMemcpyDeviceToHost);
  
    for(i = 0; i < n; i++)
    {
        cout<<host_a[i]<<" + "<<host_b[i]<<" = "<<host_c[i]<<endl;
    }
 
    hipFree(devc_a);
    hipFree(devc_b);
    hipFree(devc_c);
 
    free(host_a);
    free(host_b);
    free(host_c);
 
    return 0;
}

