#include "hip/hip_runtime.h"
%%cu
#include <bits/stdc++.h>
using namespace std;

__device__ int getFactorial(int n)
{
    if(n == 0 || n == 1)
      return n;
    return n * getFactorial(n-1);
}

__global__ void factorial(int *n, int * res)
{
    *res = getFactorial(*n);
}
int main()
{
	int host_n , host_res;
  int *devc_n, *devc_res;
  host_n = 10;
  hipMalloc(&devc_n, sizeof(int));
  hipMalloc(&devc_res, sizeof(int));
 
  hipMemcpy(devc_n, &host_n, sizeof(int), hipMemcpyHostToDevice);
 
  factorial<<<1,1>>>(devc_n, devc_res);
 
  hipMemcpy(&host_res, devc_res,sizeof(int), hipMemcpyDeviceToHost);
  
  cout<<"Factorial of "<<host_n<<" is "<<host_res;
 
  hipFree(devc_n);
  hipFree(devc_res);
	return 0;
}
